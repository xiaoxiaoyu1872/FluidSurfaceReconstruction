#include "NeighborSearch.h"
#include "NeighborSearch_kernel.cu"
#include "Thrust.cuh"

NeighborSearch::NeighborSearch(GPUmemory *_gMemory, Params *_params)
{
    gMemory = _gMemory;
    params = _params;
    constantMemCopy_Grid();
}

void NeighborSearch::constantMemCopy_Grid()
{
	// checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dGridParams), &params->mGridParams, sizeof(GridParams)));
}

void NeighborSearch::BoundGridBuilding()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dGridParams), &params->mGridParams, sizeof(GridParams)));
    uint NumBoundParticles = gMemory->NumBoundParticles;

    dim3 gridDim, blockDim;
    calcGridDimBlockDim(NumBoundParticles,gridDim, blockDim);

    boundGridBuilding<<< gridDim, blockDim, 0, 0>>>
    (gMemory->dBoundParticle, 
    NumBoundParticles, 
    gMemory->dBoundGrid,
    gMemory->dSpatialGrid);
	getLastCudaError("boundGridBuilding");

    // gMemory->Memfree_bound();
    hipDeviceSynchronize();
}  


void NeighborSearch::SpatialGridBuilding()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dGridParams), &params->mGridParams, sizeof(GridParams)));
    uint NumParticles = gMemory->NumParticles;
    uint *dParticleHash;
    
	checkCudaErrors(hipMalloc((void**)&dParticleHash, NumParticles * sizeof(uint)));
   
    dim3 gridDim, blockDim;
    calcGridDimBlockDim(NumParticles,gridDim, blockDim);

    calculateHash<<< gridDim, blockDim, 0, 0>>>(dParticleHash, gMemory->dFluidParticle, NumParticles, gMemory->dSpatialGrid);
	getLastCudaError("calculateHash");
    hipDeviceSynchronize();

    ThrustSort(gMemory->dFluidParticle, dParticleHash, NumParticles);
    getLastCudaError("thrustSort");
    hipDeviceSynchronize();

    // hipEvent_t start, stop;
    // float elapsedTime = 0.0;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);

    uint memSize = sizeof(uint) * (numThreads + 1);
    calCellRange <<< gridDim, blockDim, memSize , 0>>> (gMemory->dIndexRange, NumParticles, dParticleHash, gMemory->dFluidParticle);
	getLastCudaError("calCellRange");
    hipDeviceSynchronize();

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // std::cout << "elasped time:" << elapsedTime << std::endl; 

    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    
    safeCudaFree((void**)&dParticleHash);
}  